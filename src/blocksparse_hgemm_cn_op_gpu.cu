
#include <hip/hip_runtime.h>
#if GOOGLE_CUDA

#include "ew_op_gpu.h"
#include "gpu_hmma.h"
#include <stdio.h>

#if __CUDA_ARCH__ >= 700

template <uint OP_A, bool GATED>
__global__ void __launch_bounds__(128) hgemm_blocksparse_32x64x32_xn_sdd(
    const uint2* __restrict__ Lut,
    const float* __restrict__ Gate,
    const ehalf* __restrict__ A,
    const ehalf* __restrict__ B,
          ehalf*              C,
    uint* Lock, uint locks, uint N)
{
    const uint stdA = 48;
    const uint stdB = 80;
    const uint stdC = 132;

    __shared__ ehalf hShare[(stdA + stdB)*32];
    float* fShare = (float*)hShare;
    uint2* LutOffsets = (uint2*)&hShare[(stdA + stdB)*32];

    uint tid   = threadIdx.x;
    uint idx_N = blockIdx.x;
    uint idx_L = blockIdx.y;

    uint4 lut_head = ((const uint4*)Lut)[idx_L];
    uint lut_offset = lut_head.x;
    uint lut_size   = lut_head.y;
    uint idx_K      = lut_head.z;
    uint idx_Lock   = lut_head.w;

    uint txb = tid % 8;
    uint tyb = tid / 8;

    if (lut_size > 0)
    {
        uint* Gates = (uint*)&LutOffsets[lut_size];

        // prefetch the lut and gate data into shared
        Lut += lut_offset;
        #pragma unroll 1
        for (uint i = tid; i < lut_size; i += 128)
        {
            uint2 entry = Lut[i];

            if (GATED)
            {
                float gate = Gate[entry.y];
                uint gate2;
                asm("{                         \n\t"
                    ".reg .f16 gate;           \n\t"
                    "cvt.rn.f16.f32 gate, %1;  \n\t"
                    "mov.b32 %0, {gate, gate}; \n\t"
                    "}" : "=r"(gate2) : "f"(gate));
                Gates[i] = gate2;
            }
            else
                Gates[i] = 1;

            entry.y *= 32*32;
            entry.x *= N*32;
            LutOffsets[i] = entry;
        }
        __syncthreads();

        uint txa = tid % 4;
        uint tya = tid / 4;

        uint storA = tya*stdA + txa*8;
        uint storB = tyb*stdB + txb*8 + stdA*32;

        uint loadA = fragmentA<OP_A,M16N16K16>::get_idx(tid, stdA, (tid & 64)*(OP_A == OP_N ? 1 : stdA)*16/64);
        uint loadB = fragmentB<OP_N,M16N16K16>::get_idx(tid, stdB, (tid & 64)*stdB*16/64 + (tid & 32) + stdA*32);

        uint       n = idx_N*64 + txb*8;
        uint offsetA = tid*8;
        uint offsetB = tyb*N + n;

        asm(".reg .pred pn;\n\tsetp.lt.u32 pn, %0, %1;" :: "r"(n), "r"(N)); // n < N
        asm("mov.b32 %0, %0;" : "+r"(loadA) : );
        asm("mov.b32 %0, %0;" : "+r"(loadB) : );
        asm("mov.b32 %0, %0;" : "+r"(offsetA) : );
        asm("mov.b32 %0, %0;" : "+r"(offsetB) : );

        fragmentC<OP_A,OP_N,M16N16K16> fragC[2][2];

        int idx_lut = 0;
        #pragma unroll 1
        do
        {
            uint gate = Gates[idx_lut];

            if (gate != 0)
            {
                uint2 entry = LutOffsets[idx_lut];

                uint4 a00 = load_half8(A + (entry.y + offsetA));
                uint4 b00, b16;

                asm("mov.u32 %0, 0;\n\t"
                    "mov.u32 %1, 0;\n\t"
                    "mov.u32 %2, 0;\n\t"
                    "mov.u32 %3, 0;\n\t"
                    "mov.u32 %4, 0;\n\t"
                    "mov.u32 %5, 0;\n\t"
                    "mov.u32 %6, 0;\n\t"
                    "mov.u32 %7, 0;\n\t"
                    "@pn ld.global.nc.v4.u32 {%0, %1, %2, %3}, [%8];\n\t"
                    "@pn ld.global.nc.v4.u32 {%4, %5, %6, %7}, [%9];\n\t" :
                    "=r"(b00.x), "=r"(b00.y), "=r"(b00.z), "=r"(b00.w),
                    "=r"(b16.x), "=r"(b16.y), "=r"(b16.z), "=r"(b16.w) :
                    "l"(B + (entry.x + offsetB + N* 0)),
                    "l"(B + (entry.x + offsetB + N*16)));

                if (GATED)
                {
                    asm("mul.rn.f16x2 %0, %0, %1;" : "+r"(a00.x) : "r"(gate));
                    asm("mul.rn.f16x2 %0, %0, %1;" : "+r"(a00.y) : "r"(gate));
                    asm("mul.rn.f16x2 %0, %0, %1;" : "+r"(a00.z) : "r"(gate));
                    asm("mul.rn.f16x2 %0, %0, %1;" : "+r"(a00.w) : "r"(gate));
                }
                __syncthreads();
                *(uint4*)&hShare[storA          ] = a00;
                *(uint4*)&hShare[storB +  0*stdB] = b00;
                *(uint4*)&hShare[storB + 16*stdB] = b16;
                __syncthreads();

                fragmentA<OP_A,M16N16K16> fragA[2];
                fragmentB<OP_N,M16N16K16> fragB[2];
                for (int i = 0; i < 2; i++)
                {
                    fragA[i].load(hShare, loadA + (OP_A == OP_N ? stdA : 1)*i*16, stdA);
                    fragB[i].load(hShare, loadB + i*16, stdB);
                }
                for (int i = 0; i < 2; i++)
                    for (int j = 0; j < 2; j++)
                        fragC[i][j].mma_sync(fragA[i], fragB[j]);
            }

        } while (++idx_lut < lut_size);

        asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid  ) :);
        asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(idx_N) :);

        uint txc = tid % 16;
        uint tyc = tid / 16;

        n = idx_N*64 + txc*4;
        uint loadC   = tyc*stdC + txc*4;
        uint storC   = fragmentC<OP_A,OP_N,M16N16K16>::get_idx(tid, stdC, tid & 96);
        uint offsetC = (idx_K*32 + tyc)*N + n;

        if (idx_Lock == 0)
        {
            for (int i = 0; i < 2; i++)
            {
                __syncthreads();
                for (int j = 0; j < 2; j++)
                    fragC[i][j].store(fShare, storC + j*16, stdC);
                __syncthreads();

                if (n < N)
                    for (int j = 0; j < 2; j++)
                        store_half4(C + (offsetC + N*(j*8 + i*16)), to_half4(ew_add(
                            *(float4*)&fShare[loadC + stdC*j*8 +  0],
                            *(float4*)&fShare[loadC + stdC*j*8 + 64])));
            }
        }
        else
        {
            Lock += idx_N*locks + idx_Lock - 1;

            // Critial Section
            if (tid == 0)
                while (atomicCAS(Lock, 0, 1) != 0);
            __syncthreads();

            uint* Count   = Lock + locks * gridDim.x;
            uint  count   = *Count;
            __syncthreads();

            if (count == 0)
            {
                if (tid == 0)
                    *Count = 1;

                // first block to get here just writes out to init the memory
                for (int i = 0; i < 2; i++)
                {
                    __syncthreads();
                    for (int j = 0; j < 2; j++)
                        fragC[i][j].store(fShare, storC + j*16, stdC);
                    __syncthreads();

                    if (n < N)
                        for (int j = 0; j < 2; j++)
                            store_half4(C + (offsetC + N*(j*8 + i*16)), to_half4(ew_add(
                                *(float4*)&fShare[loadC + stdC*j*8 +  0],
                                *(float4*)&fShare[loadC + stdC*j*8 + 64])));
                }

                __threadfence();
                __syncthreads();

                if (tid == 0)
                    atomicExch(Lock, 0);
                // End Critial Section
            }
            else
            {
                txc = tid % 32;
                tyc = tid / 32;

                n       = idx_N*64 + txc*2;
                loadC   = tyc*stdC + txc*2;
                offsetC = (idx_K*32 + tyc)*N + n;

                // subsequent blocks must accumulate
                for (int i = 0; i < 2; i++)
                {
                    __syncthreads();
                    for (int j = 0; j < 2; j++)
                        fragC[i][j].store(fShare, storC + j*16, stdC);
                    __syncthreads();

                    if (n < N)
                        for (int j = 0; j < 4; j++)
                            reduce_half2(C + (offsetC + N*(j*4 + i*16)), to_half2(ew_add(
                                *(float2*)&fShare[loadC + stdC*j*4 +  0],
                                *(float2*)&fShare[loadC + stdC*j*4 + 64])));
                }

                __threadfence();
                __syncthreads();

                if (tid == 0)
                    atomicExch(Lock, 0);
                // End Critial Section
            }
        }
    }
    else
    {
        uint n = idx_N*64 + txb*8;
        uint offsetC = (idx_K*32 + tyb)*N + n;

        if (n < N)
        {
            zero_half8(C + (offsetC + N *0));
            zero_half8(C + (offsetC + N*16));
        }
    }
}

template <uint OP_A, bool GATED>
__global__ void __launch_bounds__(64) hgemm_blocksparse_16x64x16_xn_sdd(
    const uint2* __restrict__ Lut,
    const float* __restrict__ Gate,
    const ehalf* __restrict__ A,
    const ehalf* __restrict__ B,
          ehalf*              C,
    uint* Lock, uint locks, uint N)
{
    const uint stdA = 16;
    const uint stdB = 80;
    const uint stdC = 68;

    __shared__ ehalf hShare[(stdA + stdB)*16];
    uint2* LutOffsets = (uint2*)&hShare[(stdA + stdB)*16];

    uint tid   = threadIdx.x;
    uint idx_N = blockIdx.x;
    uint idx_L = blockIdx.y;

    uint4 lut_head = ((const uint4*)Lut)[idx_L];
    uint lut_offset = lut_head.x;
    uint lut_size   = lut_head.y;
    uint idx_K      = lut_head.z;
    uint idx_Lock   = lut_head.w;

    uint txb = tid % 8;
    uint tyb = tid / 8;

    if (lut_size > 0)
    {
        uint* Gates = (uint*)&LutOffsets[lut_size];

        // prefetch the lut and gate data into shared
        Lut += lut_offset;
        #pragma unroll 1
        for (uint i = tid; i < lut_size; i += 64)
        {
            uint2 entry = Lut[i];

            if (GATED)
            {
                float gate = Gate[entry.y];
                uint gate2;
                asm("{                         \n\t"
                    ".reg .f16 gate;           \n\t"
                    "cvt.rn.f16.f32 gate, %1;  \n\t"
                    "mov.b32 %0, {gate, gate}; \n\t"
                    "}" : "=r"(gate2) : "f"(gate));
                Gates[i] = gate2;
            }
            else
                Gates[i] = 1;

            entry.y *= 16*16;
            entry.x *= N*16;
            LutOffsets[i] = entry;
        }
        __syncthreads();

        uint txa = tid % 4;
        uint tya = tid / 4;

        uint storA = tya*stdA + txa*4;
        uint storB = tyb*stdB + txb*8 + 16*stdA;

        uint loadA = fragmentA<OP_A,M16N16K16>::get_idx(tid, stdA);
        uint loadB = fragmentB<OP_N,M16N16K16>::get_idx(tid, stdB, 16*stdA + (tid & 32));

        uint       n = idx_N*64 + txb*8;
        uint offsetA = tid*4;
        uint offsetB = tyb*N + n;

        asm(".reg .pred pn;\n\tsetp.lt.u32 pn, %0, %1;" :: "r"(n), "r"(N)); // n < N
        asm("mov.b32 %0, %0;" : "+r"(loadA) : );
        asm("mov.b32 %0, %0;" : "+r"(loadB) : );
        asm("mov.b32 %0, %0;" : "+r"(offsetA) : );
        asm("mov.b32 %0, %0;" : "+r"(offsetB) : );

        fragmentC<OP_A,OP_N,M16N16K16> fragC[2];

        int idx_lut = 0;
        #pragma unroll 1
        do
        {
            uint gate = Gates[idx_lut];

            if (gate != 0)
            {
                uint2 entry = LutOffsets[idx_lut];

                uint2 a0 = load_half4(A + (entry.y + offsetA));
                uint4 b0, b8;

                asm("mov.u32 %0, 0;\n\t"
                    "mov.u32 %1, 0;\n\t"
                    "mov.u32 %2, 0;\n\t"
                    "mov.u32 %3, 0;\n\t"
                    "mov.u32 %4, 0;\n\t"
                    "mov.u32 %5, 0;\n\t"
                    "mov.u32 %6, 0;\n\t"
                    "mov.u32 %7, 0;\n\t"
                    "@pn ld.global.nc.v4.u32 {%0, %1, %2, %3}, [%8];\n\t"
                    "@pn ld.global.nc.v4.u32 {%4, %5, %6, %7}, [%9];\n\t" :
                    "=r"(b0.x), "=r"(b0.y), "=r"(b0.z), "=r"(b0.w),
                    "=r"(b8.x), "=r"(b8.y), "=r"(b8.z), "=r"(b8.w) :
                    "l"(B + (entry.x + offsetB + N*0)),
                    "l"(B + (entry.x + offsetB + N*8)));

                if (GATED)
                {
                    asm("mul.rn.f16x2 %0, %0, %1;" : "+r"(a0.x) : "r"(gate));
                    asm("mul.rn.f16x2 %0, %0, %1;" : "+r"(a0.y) : "r"(gate));
                }
                __syncthreads();
                *(uint2*)&hShare[storA] = a0;
                *(uint4*)&hShare[storB + 0*stdB] = b0;
                *(uint4*)&hShare[storB + 8*stdB] = b8;
                __syncthreads();

                fragmentA<OP_A,M16N16K16> fragA;
                fragmentB<OP_N,M16N16K16> fragB;

                fragA.load(hShare, loadA, stdA);
                #pragma unroll
                for (int j = 0; j < 2; j++)
                {
                    fragB.load(hShare, loadB + j*16, stdB);

                    fragC[j].mma_sync(fragA, fragB);
                }
            }

        } while (++idx_lut < lut_size);

        // allow assembler to forget these registers in the main loop
        asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid  ) :);
        asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(idx_N) :);

         // use thread stride of 4 to allow use of shared stride of 68
        // which minimizes shared bank conflicts on write.
        uint txc = tid % 16;
        uint tyc = tid / 16;

        n = idx_N*64 + txc*4;
        uint loadC   = tyc*stdC + txc*4;
        uint storC   = fragmentC<OP_A,OP_N,M16N16K16>::get_idx(tid, stdC, tid & 32);
        uint offsetC = (idx_K*16 + tyc)*N + n;

        __syncthreads();
        for (int j = 0; j < 2; j++)
            fragC[j].store(hShare, storC + j*16, stdC);
        __syncthreads();

        if (idx_Lock == 0)
        {
            // no lock needed just write out the results
            for (uint i = 0; i < 4; i++)
                if (n < N)
                    store_half4(C + (offsetC + N*i*4), *(uint2*)&hShare[loadC + stdC*i*4]);
        }
        else
        {
            Lock += idx_N*locks + idx_Lock - 1;

            // Critial Section
            if (tid == 0)
                while (atomicCAS(Lock, 0, 1) != 0);
            __syncthreads();

            uint* Count   = Lock + locks * gridDim.x;
            uint  count   = *Count;
            __syncthreads();

            if (count == 0)
            {
                if (tid == 0)
                    *Count = 1;

                // first block to get here just writes out to init the memory
                for (uint i = 0; i < 4; i++)
                    if (n < N)
                        store_half4(C + (offsetC + N*i*4), *(uint2*)&hShare[loadC + stdC*i*4]);

                __threadfence();
                __syncthreads();

                if (tid == 0)
                    atomicExch(Lock, 0);
                // End Critial Section
            }
            else
            {
                txc = tid % 32;
                tyc = tid / 32;

                n       = idx_N*64 + txc*2;
                loadC   = tyc*stdC + txc*2;
                offsetC = (idx_K*16 + tyc)*N + n;

                // subsequent blocks must accumulate
                for (uint i = 0; i < 8; i++)
                    if (n < N)
                        reduce_half2(C + (offsetC + N*i*2), *(uint*)&hShare[loadC + stdC*i*2]);

                __threadfence();
                __syncthreads();

                if (tid == 0)
                    atomicExch(Lock, 0);
                // End Critial Section
            }
        }
    }
    else
    {
        uint n = idx_N*64 + txb*8;
        C += (idx_K*16 + tyb)*N + n;

        if (n < N)
        {
            zero_half8(C + N*0);
            zero_half8(C + N*8);
        }
    }
}

template <uint OP_A, bool GATED>
__global__ void __launch_bounds__(64) hgemm_blocksparse_8x64x8_xn_sdd(
    const uint2* __restrict__ Lut,
    const float* __restrict__ Gate,
    const ehalf* __restrict__ A,
    const ehalf* __restrict__ B,
          ehalf*              C,
    uint* Lock, uint locks, uint N)
{
    const uint stdA = 8;
    const uint stdB = 80;
    const uint stdC = 68;

    __shared__ ehalf hShare[(stdA + stdB)*16];
    uint2* LutOffsets = (uint2*)&hShare[(stdA + stdB)*16];

    uint tid   = threadIdx.x;
    uint idx_N = blockIdx.x;
    uint idx_L = blockIdx.y;

    uint4 lut_head = ((const uint4*)Lut)[idx_L];
    uint lut_offset = lut_head.x;
    uint lut_size   = lut_head.y;
    uint idx_K      = lut_head.z;
    uint idx_Lock   = lut_head.w;

    if (lut_size > 0)
    {
        uint* Gates = (uint*)&LutOffsets[lut_size];

        // prefetch the lut and gate data into shared
        Lut += lut_offset;
        #pragma unroll 1
        for (uint i = tid; i < lut_size; i += 64)
        {
            uint2 entry = Lut[i];

            if (GATED)
            {
                float gate = Gate[entry.y];
                uint gate2;
                asm("{                         \n\t"
                    ".reg .f16 gate;           \n\t"
                    "cvt.rn.f16.f32 gate, %1;  \n\t"
                    "mov.b32 %0, {gate, gate}; \n\t"
                    "}" : "=r"(gate2) : "f"(gate));
                Gates[i] = gate2;
            }
            else
                Gates[i] = 1;

            entry.y *= 8*8; // 64 entries of A per block
            entry.x *= N*8; // 8 lines of B per block
            LutOffsets[i] = entry;
        }
        if (tid == 0)
            Gates[lut_size] = 0; // needed if lut_size is odd

        __syncthreads();

        uint t32 = tid & 32;
        uint t31 = tid & 31;
        uint txb = tid % 8;
        uint tyb = t31 / 8;

        uint storA = tid*2;
        uint storB = tyb*stdB + txb*8 + t32*20 + 16*stdA;

        uint loadA = fragmentA<OP_A,M8N32K16>::get_idx(tid, stdA);
        uint loadB = fragmentB<OP_N,M8N32K16>::get_idx(tid, stdB, t32 + 16*stdA);

        uint       n = idx_N*64 + txb*8;
        uint offsetA = t31*2;
        uint offsetB = tyb*N + n;

        fragmentC<OP_A,OP_N,M8N32K16> fragC;

        uint idx_lut   = t32 / 32;
        uint idx_lut2  = 0;
        uint lut_size2 = (lut_size + 1)/2;

        asm(".reg .pred pn;\n\tsetp.lt.u32 pn, %0, %1;" :: "r"(n), "r"(N)); // n < N
        asm("mov.b32 %0, %0;" : "+r"(loadA) : );
        asm("mov.b32 %0, %0;" : "+r"(loadB) : );
        asm("mov.b32 %0, %0;" : "+r"(offsetA) : );
        asm("mov.b32 %0, %0;" : "+r"(offsetB) : );

        #pragma unroll 1
        do
        {
            uint  a0 = 0;
            uint4 b0 = {0};
            uint4 b4 = {0};

            uint gate = Gates[idx_lut];

            // if the gate is zero just skip over memory loads
            // we compute 2 blocks per loop so it's easier to just always do the mma math
            if (gate != 0)
            {
                uint2 entry = LutOffsets[idx_lut];
                a0 = load_half2(A + (entry.y + offsetA));

                asm("@pn ld.global.nc.v4.u32 {%0, %1, %2, %3}, [%8];\n\t"
                    "@pn ld.global.nc.v4.u32 {%4, %5, %6, %7}, [%9];\n\t" :
                    "=r"(b0.x), "=r"(b0.y), "=r"(b0.z), "=r"(b0.w),
                    "=r"(b4.x), "=r"(b4.y), "=r"(b4.z), "=r"(b4.w) :
                    "l"(B + (entry.x + offsetB + N*0)),
                    "l"(B + (entry.x + offsetB + N*4)));

                if (GATED)
                    asm("mul.rn.f16x2 %0, %0, %1;" : "+r"(a0) : "r"(gate));
            }

            // if (OP_A == OP_T)
            //     printf("%d %2d A:%08x B: %08x %08x %08x %08x %08x %08x %08x %08x\n", idx_K, tid, a0, b0.x,b0.y,b0.z,b0.w, b4.x,b4.y,b4.z,b4.w);

            __syncthreads();
            *(uint* )&hShare[storA         ] = a0;
            *(uint4*)&hShare[storB + 0*stdB] = b0;
            *(uint4*)&hShare[storB + 4*stdB] = b4;
            __syncthreads();

            fragmentA<OP_A,M8N32K16> fragA;
            fragmentB<OP_N,M8N32K16> fragB;

            fragA.load(hShare, loadA, stdA);
            fragB.load(hShare, loadB, stdB);

            // if (OP_A == OP_T)
            //     printf("%d %2d A:%08x %08x %08x %08x %08x %08x %08x %08x B:%08x %08x %08x %08x %08x %08x %08x %08x\n", idx_K, tid,
            //         fragA.x[0], fragA.x[1], fragA.x[2], fragA.x[3], fragA.x[4], fragA.x[5], fragA.x[6], fragA.x[7],
            //         fragB.x[0], fragB.x[1], fragB.x[2], fragB.x[3], fragB.x[4], fragB.x[5], fragB.x[6], fragB.x[7]);

            fragC.mma_sync(fragA, fragB);

            idx_lut += 2;

        } while (++idx_lut2 < lut_size2);

        // allow assembler to forget these registers in the main loop
        asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid  ) :);
        asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(idx_N) :);

        // use thread stride of 4 to allow use of shared stride of 68
        // which minimizes shared bank conflicts on write.
        uint txc = tid % 16;
        uint tyc = tid / 16;

        n = idx_N*64 + txc*4;
        uint loadC   = tyc*stdC + txc*4;
        uint storC   = fragmentC<OP_A,OP_N,M8N32K16>::get_idx(tid, stdC, tid & 32);
        uint offsetC = (idx_K*8 + tyc)*N + n;

        // if (OP_A == OP_T)
        //     printf("%d %d %2d %.0f %.0f %.0f %.0f %.0f %.0f %.0f %.0f\n", idx_K, idx_Lock, tid, fragC.x[0], fragC.x[1], fragC.x[2], fragC.x[3], fragC.x[4], fragC.x[5], fragC.x[6], fragC.x[7]);

        __syncthreads();
        fragC.store(hShare, storC, stdC);
        __syncthreads();

        if (idx_Lock == 0)
        {
            // no lock needed just write out the results
            for (uint i = 0; i < 2; i++)
                if (n < N)
                    store_half4(C + (offsetC + N*i*4), *(uint2*)&hShare[loadC + stdC*i*4]);
        }
        else
        {
            Lock += idx_N*locks + idx_Lock - 1;

            // Critial Section
            if (tid == 0)
                while (atomicCAS(Lock, 0, 1) != 0);
            __syncthreads();

            uint* Count   = Lock + locks * gridDim.x;
            uint  count   = *Count;
            __syncthreads();

            if (count == 0)
            {
                if (tid == 0)
                    *Count = 1;

                // first block to get here just writes out to init the memory
                for (uint i = 0; i < 2; i++)
                    if (n < N)
                        store_half4(C + (offsetC + N*i*4), *(uint2*)&hShare[loadC + stdC*i*4]);

                __threadfence();
                __syncthreads();

                if (tid == 0)
                    atomicExch(Lock, 0);
                // End Critial Section
            }
            else
            {
                txc = tid % 32;
                tyc = tid / 32;

                n       = idx_N*64 + txc*2;
                loadC   = tyc*stdC + txc*2;
                offsetC = (idx_K*8 + tyc)*N + n;

                // subsequent blocks must accumulate
                for (uint i = 0; i < 4; i++)
                    if (n < N)
                        reduce_half2(C +(offsetC + N*i*2), *(uint*)&hShare[loadC + stdC*i*2]);

                __threadfence();
                __syncthreads();

                if (tid == 0)
                    atomicExch(Lock, 0);
                // End Critial Section
            }
        }
    }
    else // lut_size == 0
    {
        uint txc = tid % 8;
        uint tyc = tid / 8;

        uint n       = idx_N*64 + txc*8;
        uint offsetC = (idx_K*8 + tyc)*N + n;

        if (n < N)
            zero_half8(C + offsetC);
    }
}

template <bool N64, bool GATED>
__global__ void __launch_bounds__(128) hgemm_blocksparse_32x32x64_nt_dds(
    struct plist8<ehalf> A,
    struct plist8<ehalf> B,
    ehalf*               C,
    const uint2* __restrict__ Lut,
    const float* __restrict__ Gate,
    uint params8, uint N, uint loops, uint accumulate)
{
    const uint stdAB = 72;
    const uint stdC  = 132;

    __shared__ ehalf hShare[stdAB*2*32];
    float* fShare = (float*)hShare;

    uint tid = threadIdx.x;
    uint bid = blockIdx.x;

    float gate = GATED ? Gate[bid] : 1.0f;

    if (gate != 0.0f)
    {
        uint2 lut_head = Lut[bid];

        uint tx = tid % 8;
        uint ty = tid / 8;
        uint n0 = tx  * 8;

        uint idx_A = lut_head.x;
        uint idx_B = lut_head.y;
        uint offsetA0 = (idx_A*32 + ty)*N + n0;
        uint offsetB0 = (idx_B*32 + ty)*N + n0;
        uint storAB = ty*stdAB + n0;

        uint loadA = fragmentA<OP_N,M16N16K16>::get_idx(tid, stdAB, (tid & 96)/2);
        uint loadB = fragmentB<OP_T,M16N16K16>::get_idx(tid, stdAB, (tid & 96)/2 + stdAB*32);

        fragmentC<OP_N,OP_T,M16N16K16> fragC[2][2];

        int p8 = 0;
        #pragma unroll 1
        do
        {
            const ehalf* A0;
            const ehalf* B0;
            asm("ld.param.u64 %0, [%2 + 0x160];\n\t"
                "ld.param.u64 %1, [%2 + 0x1a0];"
                : "=l"(A0), "=l"(B0) : "r"(p8));
            p8 += 8;

            uint offsetA = offsetA0;
            uint offsetB = offsetB0;
            uint n       = n0;
            uint loop    = 0;

            #pragma unroll 1
            do
            {
                asm volatile (".pragma \"nounroll\";"::); // ptxas, don't get clever

                uint4 a00 = {0}, a16 = {0};
                uint4 b00 = {0}, b16 = {0};
                if (N64 || n < N)
                {
                    a00 = load_half8(A0 + (offsetA + N*00));
                    a16 = load_half8(A0 + (offsetA + N*16));
                    b00 = load_half8(B0 + (offsetB + N*00));
                    b16 = load_half8(B0 + (offsetB + N*16));
                }
                offsetA += 64;
                offsetB += 64;
                if (!N64)
                    n += 64;

                __syncthreads();
                *(uint4*)&hShare[storAB +  0*stdAB +  0*stdAB] = a00;
                *(uint4*)&hShare[storAB + 16*stdAB +  0*stdAB] = a16;
                *(uint4*)&hShare[storAB +  0*stdAB + 32*stdAB] = b00;
                *(uint4*)&hShare[storAB + 16*stdAB + 32*stdAB] = b16;
                __syncthreads();

                fragmentA<OP_N,M16N16K16> fragA[2];
                fragmentB<OP_T,M16N16K16> fragB[2];
                for (int i = 0; i < 2; i++)
                {
                    fragA[i].load(hShare, loadA + stdAB*i*16, stdAB);
                    fragB[i].load(hShare, loadB + stdAB*i*16, stdAB);
                }
                for (int i = 0; i < 2; i++)
                    for (int j = 0; j < 2; j++)
                        fragC[i][j].mma_sync(fragA[i], fragB[j]);

            } while (++loop < loops);

        } while (p8 < params8);

        asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid) :);
        asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(bid) :);

        uint storC = fragmentC<OP_N,OP_T,M16N16K16>::get_idx(tid, stdC, (tid & 96));

        if (accumulate)
        {
            tx = tid % 16;
            ty = tid / 16;
            uint loadC   = ty*stdC + tx*2;
            uint offsetC = bid*32*32 + tid*2;

            for (int i = 0; i < 2; i++)
            {
                __syncthreads();
                for (int j = 0; j < 2; j++)
                    fragC[i][j].store(fShare, storC + j*16, stdC);
                __syncthreads();


                for (uint j = 0; j < 2; j++)
                {
                    float2 sum2 = ew_add(
                        ew_add(
                            *(float2*)&fShare[loadC + j*8*stdC +  0],
                            *(float2*)&fShare[loadC + j*8*stdC + 32]),
                        ew_add(
                            *(float2*)&fShare[loadC + j*8*stdC + 64],
                            *(float2*)&fShare[loadC + j*8*stdC + 96]));

                    reduce_half2(C + offsetC + i*4*128 + j*2*128, to_half2(sum2));

                }
            }

        }
        else
        {
            tx = tid % 8;
            ty = tid / 8;
            uint loadC   = ty*stdC + tx*4;
            uint offsetC = bid*32*32 + tid*4;

            for (int i = 0; i < 2; i++)
            {
                __syncthreads();
                for (int j = 0; j < 2; j++)
                    fragC[i][j].store(fShare, storC + j*16, stdC);
                __syncthreads();

                float4 sum4 = ew_add(
                    ew_add(
                        *(float4*)&fShare[loadC +  0],
                        *(float4*)&fShare[loadC + 32]),
                    ew_add(
                        *(float4*)&fShare[loadC + 64],
                        *(float4*)&fShare[loadC + 96]));

                store_half4(C + offsetC + i*4*128, to_half4(sum4));
            }
        }
    }
    else if (!accumulate) // gate == 0
        zero_half8(C + (bid*32*32 + tid*8));
}

template <bool N64, bool GATED>
__global__ void __launch_bounds__(64) hgemm_blocksparse_16x16x64_nt_dds(
    struct plist8<ehalf> A,
    struct plist8<ehalf> B,
    ehalf*               C,
    const uint2* __restrict__ Lut,
    const float* __restrict__ Gate,
    uint params8, uint N, uint loops, uint accumulate)
{
    const uint stdAB = 72;
    const uint stdC  = 48;

    __shared__ ehalf hShare[stdAB*2*16];
    float* fShare = (float*)hShare;

    uint tid = threadIdx.x;
    uint bid = blockIdx.x;

    float gate = GATED ? Gate[bid] : 1.0f;

    if (gate != 0.0f)
    {
        uint2 lut_head = Lut[bid];

        uint tx = tid % 8;
        uint ty = tid / 8;
        uint n0 = tx  * 8;

        uint idx_A = lut_head.x;
        uint idx_B = lut_head.y;
        uint offsetA0 = (idx_A*16 + ty)*N + n0;
        uint offsetB0 = (idx_B*16 + ty)*N + n0;
        uint storAB = ty*stdAB + n0;
        uint loadA = fragmentA<OP_N,M16N16K16>::get_idx(tid, stdAB, (tid & 32));
        uint loadB = fragmentB<OP_T,M16N16K16>::get_idx(tid, stdAB, (tid & 32) + 16*stdAB);

        fragmentC<OP_N,OP_T,M16N16K16> fragC;

        int p8 = 0;
        #pragma unroll 1
        do
        {
            const ehalf* A0;
            const ehalf* B0;
            asm("ld.param.u64 %0, [%2 + 0x160];\n\t"
                "ld.param.u64 %1, [%2 + 0x1a0];"
                : "=l"(A0), "=l"(B0) : "r"(p8));
            p8 += 8;

            uint offsetA = offsetA0;
            uint offsetB = offsetB0;
            uint n       = n0;
            uint loop    = 0;

            #pragma unroll 1
            do
            {
                asm volatile (".pragma \"nounroll\";"::); // ptxas, don't get clever

                uint4 a0 = {0}, a8 = {0};
                uint4 b0 = {0}, b8 = {0};
                if (N64 || n < N)
                {
                    a0 = load_half8(A0 + (offsetA + N*0));
                    a8 = load_half8(A0 + (offsetA + N*8));
                    b0 = load_half8(B0 + (offsetB + N*0));
                    b8 = load_half8(B0 + (offsetB + N*8));
                }
                offsetA += 64;
                offsetB += 64;
                if (!N64)
                    n += 64;

                __syncthreads();
                *(uint4*)&hShare[storAB + 0*stdAB +  0*stdAB] = a0;
                *(uint4*)&hShare[storAB + 8*stdAB +  0*stdAB] = a8;
                *(uint4*)&hShare[storAB + 0*stdAB + 16*stdAB] = b0;
                *(uint4*)&hShare[storAB + 8*stdAB + 16*stdAB] = b8;
                __syncthreads();

                fragmentA<OP_N,M16N16K16> fragA;
                fragmentB<OP_T,M16N16K16> fragB;
                #pragma unroll
                for (uint j = 0; j < 2; j++)
                {
                    fragA.load(hShare, loadA + j*16, stdAB);
                    fragB.load(hShare, loadB + j*16, stdAB);

                    fragC.mma_sync(fragA, fragB);
                }

            } while (++loop < loops);

        } while (p8 < params8);

        asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid) :);
        asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(bid) :);

        uint storC = fragmentC<OP_N,OP_T,M16N16K16>::get_idx(tid, stdC, (tid & 32)/2);

        __syncthreads();
        fragC.store(fShare, storC, stdC);
        __syncthreads();

        if (accumulate)
        {
            tx = tid % 8;
            ty = tid / 8;
            uint loadC   = ty*stdC + tx*2;
            uint offsetC = bid*16*16 + tid*2;

            for (uint i = 0; i < 2; i++)
                reduce_half2(C + offsetC + i*2*64, to_half2(ew_add(
                    *(float2*)&fShare[loadC + i*8*stdC +  0],
                    *(float2*)&fShare[loadC + i*8*stdC + 16])));
        }
        else
        {
            tx = tid % 4;
            ty = tid / 4;
            uint loadC   = ty*stdC + tx*4;
            uint offsetC = bid*16*16 + tid*4;

            store_half4(C + offsetC, to_half4(ew_add(
                *(float4*)&fShare[loadC +  0],
                *(float4*)&fShare[loadC + 16])));
        }
    }
    else if (!accumulate) // gate == 0
        zero_half4(C + (bid*16*16 + tid*4));
}

template <bool N64, bool GATED>
__global__ void __launch_bounds__(32) hgemm_blocksparse_8x8x64_nt_dds(
    struct plist8<ehalf> A,
    struct plist8<ehalf> B,
    ehalf*               C,
    const uint2* __restrict__ Lut,
    const float* __restrict__ Gate,
    uint params8, uint N, uint loops, uint accumulate)
{
    const uint stdAB = 72;
    const uint stdC  = 8;

    __shared__ ehalf hShare[stdAB*8*2];
    float* fShare = (float*)hShare;

    uint tid = threadIdx.x;
    uint bid = blockIdx.x;

    float gate = GATED ? Gate[bid] : 1.0f;

    if (gate != 0.0f)
    {
        uint2 lut_head = Lut[bid];

        uint tx = tid % 8;
        uint ty = tid / 8;
        uint n0 = tx  * 8;

        uint idx_A = lut_head.x;
        uint idx_B = lut_head.y;
        uint offsetA0 = (idx_A*8 + ty)*N + n0;
        uint offsetB0 = (idx_B*8 + ty)*N + n0;
        uint storAB = ty*stdAB + n0;
        uint loadA = fragmentA<OP_N,M8N8K16>::get_idx(tid, stdAB, 0*stdAB);
        uint loadB = fragmentB<OP_T,M8N8K16>::get_idx(tid, stdAB, 8*stdAB);

        fragmentC<OP_N,OP_T,M8N8K16> fragC;

        int p8 = 0;
        #pragma unroll 1
        do
        {
            const ehalf* A0;
            const ehalf* B0;
            asm("ld.param.u64 %0, [%2 + 0x160];\n\t"
                "ld.param.u64 %1, [%2 + 0x1a0];"
                : "=l"(A0), "=l"(B0) : "r"(p8));
            p8 += 8;

            uint offsetA = offsetA0;
            uint offsetB = offsetB0;
            uint n       = n0;
            uint loop    = 0;
            #pragma unroll 1
            do
            {
                uint4 a0 = {0}, a4 = {0};
                uint4 b0 = {0}, b4 = {0};

                if (N64 || n < N)
                {
                    a0 = load_half8(A0 + (offsetA + N*0));
                    a4 = load_half8(A0 + (offsetA + N*4));
                    b0 = load_half8(B0 + (offsetB + N*0));
                    b4 = load_half8(B0 + (offsetB + N*4));
                }
                offsetA += 64;
                offsetB += 64;
                if (!N64)
                    n += 64;

                *(uint4*)&hShare[storAB + 0*stdAB + 0*stdAB] = a0;
                *(uint4*)&hShare[storAB + 4*stdAB + 0*stdAB] = a4;
                *(uint4*)&hShare[storAB + 0*stdAB + 8*stdAB] = b0;
                *(uint4*)&hShare[storAB + 4*stdAB + 8*stdAB] = b4;

                fragmentA<OP_N,M8N8K16> fragA;
                fragmentB<OP_T,M8N8K16> fragB;
                #pragma unroll
                for (uint j = 0; j < 4; j++)
                {
                    fragA.load(hShare, loadA + j*16, stdAB);
                    fragB.load(hShare, loadB + j*16, stdAB);

                    fragC.mma_sync(fragA, fragB);
                }

            } while (++loop < loops);

        } while (p8 < params8);

        asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid)   :);
        asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(bid)   :);

        uint storC = fragmentC<OP_N,OP_T,M8N8K16>::get_idx(tid, stdC);

        fragC.store(fShare, storC, stdC);

        C += bid*8*8 + tid*2;

        uint c2 = to_half2(ew_mul(*(float2*)&fShare[tid*2], gate));

        if (accumulate)
            reduce_half2(C, c2);
        else
            store_half2(C, c2);
    }
    else if (!accumulate) // gate == 0
        zero_half2(C + (bid*8*8 + tid*2));
}


#else // __CUDA_ARCH__ >= 700


template <uint OP_A, bool GATED>
__global__ void __launch_bounds__(128) hgemm_blocksparse_32x64x32_xn_sdd(
    const uint2* __restrict__ Lut,
    const float* __restrict__ Gate,
    const ehalf* __restrict__ A,
    const ehalf* __restrict__ B,
          ehalf*              C,
    uint* Lock, uint locks, uint N)
{
    *C = 0;
}
template <bool N64, bool GATED>
__global__ void __launch_bounds__(128) hgemm_blocksparse_32x32x64_nt_dds(
    struct plist8<ehalf> A,
    struct plist8<ehalf> B,
    ehalf*               C,
    const uint2* __restrict__ Lut,
    const float* __restrict__ Gate,
    uint params8, uint N, uint loops, uint accumulate)
{
    *C = 0;
}
template <uint OP_A, bool GATED>
__global__ void __launch_bounds__(64) hgemm_blocksparse_16x64x16_xn_sdd(
    const uint2* __restrict__ Lut,
    const float* __restrict__ Gate,
    const ehalf* __restrict__ A,
    const ehalf* __restrict__ B,
          ehalf*              C,
    uint* Lock, uint locks, uint N)
{
    *C = 0;
}
template <bool N64, bool GATED>
__global__ void __launch_bounds__(64) hgemm_blocksparse_16x16x64_nt_dds(
    struct plist8<ehalf> A,
    struct plist8<ehalf> B,
    ehalf*               C,
    const uint2* __restrict__ Lut,
    const float* __restrict__ Gate,
    uint params8, uint N, uint loops, uint accumulate)
{
    *C = 0;
}
template <uint OP_A, bool GATED>
__global__ void __launch_bounds__(64) hgemm_blocksparse_8x64x8_xn_sdd(
    const uint2* __restrict__ Lut,
    const float* __restrict__ Gate,
    const ehalf* __restrict__ A,
    const ehalf* __restrict__ B,
          ehalf*              C,
    uint* Lock, uint locks, uint N)
{
    *C = 0;
}
template <bool N64, bool GATED>
__global__ void __launch_bounds__(32) hgemm_blocksparse_8x8x64_nt_dds(
    struct plist8<ehalf> A,
    struct plist8<ehalf> B,
    ehalf*               C,
    const uint2* __restrict__ Lut,
    const float* __restrict__ Gate,
    uint params8, uint N, uint loops, uint accumulate)
{
    *C = 0;
}

#endif // __CUDA_ARCH__ >= 700

cudaError_t hgemm_blocksparse_xn_sdd(const ehalf* X, const ehalf* W, ehalf* Y, bsmm_params* params, uint op)
{
    dim3 grid(CEIL_DIV(params->N, 64), params->segments, 1);

    if (params->locks > 0)
        cuMemsetD32Async((CUdeviceptr)params->Lock, 0, grid.x * params->locks * 2, params->stream);

    const uint2* Lut = (const uint2*)params->Lut;
    uint* Lock       = (uint*)params->Lock;

    uint shared = params->shared + params->shared/2;

    if (params->bshift == 3)
    {
        shared += 4;
        if (params->Gate == 0)
        {
            if (op == OP_N)
                hgemm_blocksparse_8x64x8_xn_sdd<OP_N,false><<<grid,64,shared,params->stream>>>(Lut, params->Gate, W, X, Y, Lock, params->locks, params->N);
            else
                hgemm_blocksparse_8x64x8_xn_sdd<OP_T,false><<<grid,64,shared,params->stream>>>(Lut, params->Gate, W, X, Y, Lock, params->locks, params->N);
        }
        else
        {
            if (op == OP_N)
                hgemm_blocksparse_8x64x8_xn_sdd<OP_N, true><<<grid,64,shared,params->stream>>>(Lut, params->Gate, W, X, Y, Lock, params->locks, params->N);
            else
                hgemm_blocksparse_8x64x8_xn_sdd<OP_T, true><<<grid,64,shared,params->stream>>>(Lut, params->Gate, W, X, Y, Lock, params->locks, params->N);
        }
    }
    else if (params->bshift == 4)
    {
        if (params->Gate == 0)
        {
            if (op == OP_N)
                hgemm_blocksparse_16x64x16_xn_sdd<OP_N,false><<<grid,64,shared,params->stream>>>(Lut, params->Gate, W, X, Y, Lock, params->locks, params->N);
            else
                hgemm_blocksparse_16x64x16_xn_sdd<OP_T,false><<<grid,64,shared,params->stream>>>(Lut, params->Gate, W, X, Y, Lock, params->locks, params->N);
        }
        else
        {
            if (op == OP_N)
                hgemm_blocksparse_16x64x16_xn_sdd<OP_N, true><<<grid,64,shared,params->stream>>>(Lut, params->Gate, W, X, Y, Lock, params->locks, params->N);
            else
                hgemm_blocksparse_16x64x16_xn_sdd<OP_T, true><<<grid,64,shared,params->stream>>>(Lut, params->Gate, W, X, Y, Lock, params->locks, params->N);
        }
    }
    else if (params->bshift == 5)
    {
        if (params->Gate == 0)
        {
            if (op == OP_N)
                hgemm_blocksparse_32x64x32_xn_sdd<OP_N,false><<<grid,128,shared,params->stream>>>(Lut, params->Gate, W, X, Y, Lock, params->locks, params->N);
            else
                hgemm_blocksparse_32x64x32_xn_sdd<OP_T,false><<<grid,128,shared,params->stream>>>(Lut, params->Gate, W, X, Y, Lock, params->locks, params->N);
        }
        else
        {
            if (op == OP_N)
                hgemm_blocksparse_32x64x32_xn_sdd<OP_N, true><<<grid,128,shared,params->stream>>>(Lut, params->Gate, W, X, Y, Lock, params->locks, params->N);
            else
                hgemm_blocksparse_32x64x32_xn_sdd<OP_T, true><<<grid,128,shared,params->stream>>>(Lut, params->Gate, W, X, Y, Lock, params->locks, params->N);
        }
    }
    return cudaPeekAtLastError();
}
cudaError_t hgemm_blocksparse_xn_sdd(const bhalf* X, const bhalf* W, bhalf* Y, bsmm_params* params, uint op) { return cudaSuccess; }
cudaError_t hgemm_blocksparse_xn_sdd(const float* X, const float* W, float* Y, bsmm_params* params, uint op) { return cudaSuccess; }


cudaError_t hgemm_blocksparse_nt_dds(const ehalf* X, const ehalf* E, ehalf* U, bsmm_params* params)
{
    struct plist8<ehalf>* X8 = (struct plist8<ehalf>*)X;
    struct plist8<ehalf>* E8 = (struct plist8<ehalf>*)E;

    const uint2* Lut = (const uint2*)params->Lut;
    uint accumulate  = params->beta == 1.0f;
    uint pcount8     = params->pcount * 8;
    uint N           = params->N;
    uint loops       = CEIL_DIV(N, 64);
    bool k64         = (N & 63) == 0;

    dim3 grid(params->blocks, 1, 1);

    if (params->bshift == 3)
    {
        if (params->Gate == 0)
        {
            if (k64)
                hgemm_blocksparse_8x8x64_nt_dds< true,false><<<grid,32,0,params->stream>>>(*X8, *E8, U, Lut, params->Gate, pcount8, N, loops, accumulate);
            else
                hgemm_blocksparse_8x8x64_nt_dds<false,false><<<grid,32,0,params->stream>>>(*X8, *E8, U, Lut, params->Gate, pcount8, N, loops, accumulate);
        }
        else
        {
            if (k64)
                hgemm_blocksparse_8x8x64_nt_dds< true, true><<<grid,32,0,params->stream>>>(*X8, *E8, U, Lut, params->Gate, pcount8, N, loops, accumulate);
            else
                hgemm_blocksparse_8x8x64_nt_dds<false, true><<<grid,32,0,params->stream>>>(*X8, *E8, U, Lut, params->Gate, pcount8, N, loops, accumulate);
        }
    }
    else if (params->bshift == 4)
    {
        if (params->Gate == 0)
        {
            if (k64)
                hgemm_blocksparse_16x16x64_nt_dds< true,false><<<grid,64,0,params->stream>>>(*X8, *E8, U, Lut, params->Gate, pcount8, N, loops, accumulate);
            else
                hgemm_blocksparse_16x16x64_nt_dds<false,false><<<grid,64,0,params->stream>>>(*X8, *E8, U, Lut, params->Gate, pcount8, N, loops, accumulate);
        }
        else
        {
            if (k64)
                hgemm_blocksparse_16x16x64_nt_dds< true, true><<<grid,64,0,params->stream>>>(*X8, *E8, U, Lut, params->Gate, pcount8, N, loops, accumulate);
            else
                hgemm_blocksparse_16x16x64_nt_dds<false, true><<<grid,64,0,params->stream>>>(*X8, *E8, U, Lut, params->Gate, pcount8, N, loops, accumulate);
        }
    }
    else if (params->bshift == 5)
    {
        if (params->Gate == 0)
        {
            if (k64)
                hgemm_blocksparse_32x32x64_nt_dds< true,false><<<grid,128,0,params->stream>>>(*X8, *E8, U, Lut, params->Gate, pcount8, N, loops, accumulate);
            else
                hgemm_blocksparse_32x32x64_nt_dds<false,false><<<grid,128,0,params->stream>>>(*X8, *E8, U, Lut, params->Gate, pcount8, N, loops, accumulate);
        }
        else
        {
            if (k64)
                hgemm_blocksparse_32x32x64_nt_dds< true, true><<<grid,128,0,params->stream>>>(*X8, *E8, U, Lut, params->Gate, pcount8, N, loops, accumulate);
            else
                hgemm_blocksparse_32x32x64_nt_dds<false, true><<<grid,128,0,params->stream>>>(*X8, *E8, U, Lut, params->Gate, pcount8, N, loops, accumulate);
        }
    }
    return cudaPeekAtLastError();
}
cudaError_t hgemm_blocksparse_nt_dds(const bhalf* X, const bhalf* E, bhalf* U, bsmm_params* params) { return cudaSuccess; }
cudaError_t hgemm_blocksparse_nt_dds(const float* X, const float* E, float* U, bsmm_params* params) { return cudaSuccess; }

#endif // GOOGLE_CUDA
