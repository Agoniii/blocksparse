
#include <hip/hip_runtime.h>
#if GOOGLE_CUDA

#include "ew_op_gpu.h"
#include <stdio.h>

__device__ __forceinline__ uint bfe(uint val, int pos)
{
    uint bit;
    asm ("bfe.u32 %0, %1, %2, 1;" : "=r"(bit) : "r"(val), "r"(pos)  );
    return bit;
}

typedef struct __align__(8) KeyVal
{
    uint  key;
    float val;
} KeyVal;

template <typename T>
__global__ void top_k(T* Y, uint* A, const T* __restrict__ X, uint Exp, uint topK, uint K, uint rect, uint rebase)
{
    extern __shared__ KeyVal data[];

    uint tid = threadIdx.x;
    uint   n = blockIdx.x;

    uint offset = n*K + tid;

    KeyVal init;
    init.key  = tid;
    init.val  = tid < K ? load(add_ptr_u(X, offset)) : -FLT_MAX;
    data[tid] = init;

    __syncthreads();

    for (int i = 1; i <= Exp; ++i)
    {
        int j;
        #pragma unroll 1
        for (j = i - 1; j >= 5; --j)
        {
            // when the comparison stride is 32 or greater,
            // use half of warps and uniform shared memory access to make comparisons
            if (tid < blockDim.x/2)
            {
                // figure out the a and b indexes for the "butterfly" compare operation
                uint m = (tid >> j) << (j + 1);
                uint r =  tid & ((1 << j) - 1);
                uint a = m + r;
                uint b = a + (1 << j);
                bool d = bfe(a, i) != 0;

                KeyVal A = data[a];
                KeyVal B = data[b];

                if((B.val > A.val) ^ d)
                {
                    KeyVal t = A;
                    A = B;
                    B = t;
                }
                data[a] = A;
                data[b] = B;
            }
            __syncthreads();
        }

        // When the comparison stride is less than 32,
        // use all warps and shfl_xor operations to make comparisons in registers

        // Load shared to registers
        KeyVal A = data[tid];

        #pragma unroll 5
        while (j >= 0)
        {
            KeyVal B;
            B.val = shfl_xor(A.val, 1 << j);
            B.key = shfl_xor(A.key, 1 << j);
            bool d = bfe(tid, i) != bfe(tid, j--);

            // in the case of equality we want both shuffle lanes to not swap
            if(((B.val > A.val) ^ d) && B.val != A.val)
                A = B;
        }
        // Load final register values back to shared.
        data[tid] = A;

        __syncthreads();
    }
    if (rect)
    {
        // avoid extra __syncthreads by coalescing to unused shared
        float* coalesce = (float*)&data[blockDim.x];

        // Output same size as input, with zeros for non-topK values.
        // rebase sets the zero line to the min value of the topK

        KeyVal out = data[tid];
        float base = rebase ? fmaxf(data[topK-1].val, 0.0f) : 0.0f;
        float val  = tid < topK ? out.val : 0.0f;

        //if (tid == 0 && n == 0)
        //    printf("base: %f %d\n", base, data[topK-1].key);

        // apply the rectification and coalesce the output
        coalesce[out.key] = fmaxf(val, base) - base;

        __syncthreads();

        if (tid < K)
            store(add_ptr_u(Y, offset), coalesce[tid]);
    }
    else
    {
        // output just top values and their indicies.
        if (tid < topK)
        {
            KeyVal out = data[tid];
            offset = n*topK + tid;
            store(add_ptr_u(Y, offset), out.val);
            __stg(add_ptr_u(A, offset), out.key);
        }
    }
}

template <typename T>
bool TopK(CUstream stream, T* y, uint* a, const T* x, uint topK, uint N, uint K, uint rebase)
{
    uint exp;
         if (K > 512) exp = 10;
    else if (K > 256) exp =  9;
    else if (K > 128) exp =  8;
    else if (K >  64) exp =  7;
    else if (K >  32) exp =  6;
    else              exp =  5;
    uint threads = 1 << exp;
    uint shared  = threads * 16;

    top_k<T><<<N,threads,shared,stream>>>(y, a, x, exp, topK, K, a == NULL, rebase);
    return true;
}

template bool TopK<float>(CUstream stream, float* y, uint* a, const float* x, uint topK, uint N, uint K, uint rebase);
template bool TopK<ehalf>(CUstream stream, ehalf* y, uint* a, const ehalf* x, uint topK, uint N, uint K, uint rebase);
template bool TopK<bhalf>(CUstream stream, bhalf* y, uint* a, const bhalf* x, uint topK, uint N, uint K, uint rebase);


template <typename T>
__global__ void masked_top_k_softmax(T* Y, const float* __restrict__ M, const T* __restrict__ X, uint Exp, uint topK, uint D123, uint D23, uint D3, uint M1, uint M2, uint use_mask, float scale)
{
    extern __shared__ KeyVal block[];
    extern __shared__ float  stage[];

    // x: D0, D1, D2, D3
    // m:  1, D1, D2, D3
    // m:  1,  1, D2, D3
    // m:  1,  1,  1, D3
    uint tid = threadIdx.x;
    uint  d0 = blockIdx.x;
    uint  d1 = blockIdx.y;
    uint  d2 = blockIdx.z;

    uint offsetX = d0*D123 + d1*D23 + d2*D3 + tid;
    uint offsetM =           d1*M1  + d2*M2 + tid;

    M = add_ptr_u(M, offsetM);
    X = add_ptr_u(X, offsetX);

    float mask = tid < D3 ? (use_mask ? __ldg(M) : 1.0f) : 0.0f;
    float xval = mask != 0.0 ? load(X) * mask * scale : -FLT_MAX;

    KeyVal init;
    init.key   = tid;
    init.val   = xval;
    block[tid] = init;

    __syncthreads();

    for (int i = 1; i <= Exp; ++i)
    {
        int j;
        #pragma unroll 1
        for (j = i - 1; j >= 5; --j)
        {
            // when the comparison stride is 32 or greater,
            // use half of warps and uniform shared memory access to make comparisons
            if (tid < blockDim.x/2)
            {
                // figure out the a and b indexes for the "butterfly" compare operation
                uint m = (tid >> j) << (j + 1);
                uint r =  tid & ((1 << j) - 1);
                uint a = m + r;
                uint b = a + (1 << j);
                bool d = bfe(a, i) != 0;

                KeyVal A = block[a];
                KeyVal B = block[b];

                if((B.val > A.val) ^ d)
                {
                    KeyVal t = A;
                    A = B;
                    B = t;
                }
                block[a] = A;
                block[b] = B;
            }
            __syncthreads();
        }

        // When the comparison stride is less than 32,
        // use all warps and shfl_xor operations to make comparisons in registers

        // Load shared to registers
        KeyVal A = block[tid];

        #pragma unroll 5
        while (j >= 0)
        {
            KeyVal B;
            B.val = shfl_xor(A.val, 1 << j);
            B.key = shfl_xor(A.key, 1 << j);
            bool d = bfe(tid, i) != bfe(tid, j--);

            // in the case of equality we want both shuffle lanes to not swap
            if(((B.val > A.val) ^ d) && B.val != A.val)
                A = B;
        }
        // Load final register values back to shared.
        block[tid] = A;

        __syncthreads();
    }

    float* vals = &stage[blockDim.x*2];
    float* reds =  &vals[blockDim.x];

    KeyVal out = block[tid];
    float  val = 0.0f;
    if (tid < topK)
        val = expf(out.val - block[0].val);

    vals[out.key] = val;

    // reduce within warp
    #pragma unroll
    for (int i = 16; i > 0; i >>= 1)
        val += shfl_xor(val, i);

    // first thread of each warp store to shared
    if ((tid & 31) == 0)
        reds[tid/32] = val;

    __syncthreads();

    if (tid < blockDim.x/32)
    {
        // first warp loads all prior reductions
        val = reds[tid];
        // reduce within this last warp
        #pragma unroll 1
        for (int i = blockDim.x/64; i > 0; i >>= 1)
            val += shfl_xor(val, i);

        // rcp final reduction to shared
        reds[tid] = 1.0f / val;
    }
    __syncthreads();

    if (tid < D3)
        store(add_ptr_u(Y, offsetX), vals[tid] * reds[0]);
}

template <typename T>
bool MaskedTopKSoftmax(CUstream stream, T* y, const float* m, const T* x, uint topK, uint D0, uint D1, uint D2, uint D3, uint M1, uint M2, float scale)
{
    uint exp;
         if (D3 > 512) exp = 10;
    else if (D3 > 256) exp =  9;
    else if (D3 > 128) exp =  8;
    else if (D3 >  64) exp =  7;
    else if (D3 >  32) exp =  6;
    else               exp =  5;
    uint threads = 1 << exp;
    uint shared  = threads * 16;

    masked_top_k_softmax<T><<<dim3(D0,D1,D2),threads,shared,stream>>>(y, m, x, exp, topK, D1*D2*D3, D2*D3, D3, M1, M2, m != NULL, scale);
    return true;
}

template bool MaskedTopKSoftmax<float>(CUstream stream, float* y, const float* m, const float* x, uint topK, uint D0, uint D1, uint D2, uint D3, uint M1, uint M2, float scale);
template bool MaskedTopKSoftmax<ehalf>(CUstream stream, ehalf* y, const float* m, const ehalf* x, uint topK, uint D0, uint D1, uint D2, uint D3, uint M1, uint M2, float scale);
template bool MaskedTopKSoftmax<bhalf>(CUstream stream, bhalf* y, const float* m, const bhalf* x, uint topK, uint D0, uint D1, uint D2, uint D3, uint M1, uint M2, float scale);

// x *= mask * scale
// y  = exp(x - max(x)) / sum( exp(x - max(x)) )
template <typename T, int U>
__global__ void masked_softmax(
              T*              Y,
    const     T* __restrict__ X,
    const float* __restrict__ M,
    uint D123, uint D23, uint D3, uint M1, uint M2, uint use_mask, float scale, int threads_pow2)
{
    __shared__ float Max[32];
    __shared__ float Sum[32];
    // x: D0, D1, D2, D3
    // m:  1, D1, D2, D3
    // m:  1,  1, D2, D3
    // m:  1,  1,  1, D3
    uint tid = threadIdx.x;
    uint  d0 = blockIdx.x;
    uint  d1 = blockIdx.y;
    uint  d2 = blockIdx.z;

    if (blockDim.x > 32)
    {
        if (tid < 32)
        {
            // Allows non-power of 2 threads to work
            Max[tid] = -FLT_MAX;
            Sum[tid] = 0.0f;
        }
        __syncthreads();
    }

    uint ti = (tid & 0x3fe0)*U + (tid & 31);
    uint offsetX = d0*D123 + d1*D23 + d2*D3 + ti;
    uint offsetM =           d1*M1  + d2*M2 + ti;

    M = add_ptr_u(M, offsetM);
    X = add_ptr_u(X, offsetX);

    // Load mask
    float mask[U]; for (int i = 0; i < U; i++) mask[i]= 1.0f;
    if (use_mask)
    {
        for (int i = 0; i < U; i++)
        {
            mask[i] = 0.0f;
            if (ti + i*32 < D3)
                mask[i] = __ldg(M + i*32);
        }
    }
    // Load X
    float xval[U]; for (int i = 0; i < U; i++) xval[i] = -FLT_MAX;
    for (int i = 0; i < U; i++)
        if (mask[i] != 0.0 && ti + i*32 < D3)
            xval[i] = load(X, i*32) * mask[i] * scale;

    // reduce within thread
    float Xmax[U];
    for (int i = 0; i < U; i++)
        Xmax[i] = xval[i];

    for (int j = U >> 1; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
            Xmax[i] = fmaxf(Xmax[i], Xmax[i+j]);
    float xmax = Xmax[0];

    // reduce within warp
    for (int i = 16; i > 0; i >>= 1)
        xmax = fmaxf(xmax, shfl_xor(xmax, i));

    if (blockDim.x > 32)
    {
        // first thread of each warp store to shared
        if ((tid & 31) == 0)
            Max[tid/32] = xmax;
        __syncthreads();
        if (tid < 32)
        {
            // first warp loads all prior reductions
            xmax = Max[tid];
            // reduce within this last warp
            #pragma unroll 1
            for (int i = threads_pow2/64; i > 0; i >>= 1)
                xmax = fmaxf(xmax, shfl_xor(xmax, i));
            // final reduction to shared
            Max[tid] = xmax;
        }
        __syncthreads();
        xmax = Max[0];
    }
    // compute exponent of softmax
    float Xsum[U];
    for (int i = 0; i < U; i++)
        Xsum[i] = xval[i] = expf(xval[i] - xmax);

    // reduce within thread
    for (int j = U >> 1; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
            Xsum[i] = Xsum[i] + Xsum[i+j];
    float exp_sum = Xsum[0];

    // reduce within warp
    for (int i = 16; i > 0; i >>= 1)
        exp_sum += shfl_xor(exp_sum, i);

    if (blockDim.x > 32)
    {
        // first thread of each warp store to shared
        if ((tid & 31) == 0)
            Sum[tid/32] = exp_sum;
        __syncthreads();

        if (tid < 32)
        {
            // first warp loads all prior reductions
            exp_sum = Sum[tid];
            // reduce within this last warp
            #pragma unroll 1
            for (int i = threads_pow2/64; i > 0; i >>= 1)
                exp_sum += shfl_xor(exp_sum, i);
            // final reduction to shared
            Sum[tid] = exp_sum;
        }
        __syncthreads();
        exp_sum = Sum[0];
    }
    float rcp_exp_sum = 1.0f / exp_sum;

    Y = add_ptr_u(Y, offsetX);

    for (int i = 0; i < U; i++)
        store(Y, xval[i] * rcp_exp_sum, i*32, ti + i*32 < D3);
}

// x *= mask * scale
// y  = exp(x - max(x)) / sum( exp(x - max(x)) )
template <typename T>
__global__ void __launch_bounds__(32) masked_softmax2(
              T*              Y,
    const     T* __restrict__ X,
    const float* __restrict__ M,
    uint D123, uint D23, uint D3, uint M1, uint M2, uint use_mask, float scale)
{
    // x: D0, D1, D2, D3
    // m:  1, D1, D2, D3
    // m:  1,  1, D2, D3
    // m:  1,  1,  1, D3
    uint tid = threadIdx.x;
    uint  d0 = blockIdx.x;
    uint  d1 = blockIdx.y;
    uint  d2 = blockIdx.z;

    uint offsetX = d0*D123 + d1*D23 + d2*D3 + tid;
    uint offsetM =           d1*M1  + d2*M2 + tid;

    // max(x, axis-1)
    float max_x = -FLT_MAX;
    #pragma unroll 2
    for (uint d3 = tid, xi = offsetX, mi = offsetM; d3 < D3; d3 += 32, xi += 32, mi += 32)
    {
        float m = use_mask ? __ldg(add_ptr_u(M, mi)) : 1.0f;
        float x = m != 0.0 ? load(add_ptr_u(X, xi)) * m * scale : -FLT_MAX;

        max_x = fmaxf(max_x, x);
    }
    for (int i = 16; i > 0; i >>= 1)
        max_x = fmaxf(max_x, shfl_xor(max_x, i));

    float exp_sum = 0.0f;
    #pragma unroll 2
    for (uint d3 = tid, xi = offsetX, mi = offsetM; d3 < D3; d3 += 32, xi += 32, mi += 32)
    {
        float m = use_mask ? __ldg(add_ptr_u(M, mi)) : 1.0f;
        float x = m != 0.0 ? load(add_ptr_u(X, xi)) * m * scale : -FLT_MAX;

        exp_sum += expf(x - max_x);
    }
    for (int i = 16; i > 0; i >>= 1)
        exp_sum += shfl_xor(exp_sum, i);

    float rcp_exp_sum = 1.0f / exp_sum;

    #pragma unroll 2
    for (uint d3 = tid, xi = offsetX, mi = offsetM; d3 < D3; d3 += 32, xi += 32, mi += 32)
    {
        float m = use_mask ? __ldg(add_ptr_u(M, mi)) : 1.0f;
        float x = m != 0.0 ? load(add_ptr_u(X, xi)) * m * scale : -FLT_MAX;

        float y = expf(x - max_x)  * rcp_exp_sum;

        store(add_ptr_u(Y, xi), y);
    }
}

// dx = (dy - sum(dy * y, axis=-1)) * y * m * scale
template <typename T, int U>
__global__ void masked_softmax_grad(
              T*              DX,
    const     T* __restrict__ DY,
    const     T* __restrict__ Y,
    const float* __restrict__ M,
    uint D123, uint D23, uint D3, uint M1, uint M2, uint use_mask, float scale, int threads_pow2)
{
    __shared__ float Sum[32];
    // x: D0, D1, D2, D3
    // m:  1, D1, D2, D3
    // m:  1,  1, D2, D3
    // m:  1,  1,  1, D3
    uint tid = threadIdx.x;
    uint  d0 = blockIdx.x;
    uint  d1 = blockIdx.y;
    uint  d2 = blockIdx.z;

    if (blockDim.x > 32)
    {
        // Allows non-power of 2 threads to work
        if (tid < 32)
            Sum[tid] = 0.0f;
        __syncthreads();
    }

    uint ti = (tid & 0x3fe0)*U + (tid & 31);
    uint offsetY = d0*D123 + d1*D23 + d2*D3 + ti;
    uint offsetM =           d1*M1  + d2*M2 + ti;

    DY = add_ptr_u(DY, offsetY);
    Y  = add_ptr_u( Y, offsetY);
    M  = add_ptr_u( M, offsetM);

    // Load mask
    float mask[U]; for (int i = 0; i < U; i++) mask[i]= 1.0f;
    if (use_mask)
    {
        for (int i = 0; i < U; i++)
        {
            mask[i] = 0.0f;
            if (ti + i*32 < D3)
                mask[i] = __ldg(M + i*32);
        }
    }
    // Load DY
    float dy[U]; for (int i = 0; i < U; i++) dy[i]= 0.0f;
    for (int i = 0; i < U; i++)
        if (mask[i] != 0.0 && ti + i*32 < D3)
            dy[i] = load(DY, i*32);

    // Load Y
    float y[U]; for (int i = 0; i < U; i++) y[i]= 0.0f;
    for (int i = 0; i < U; i++)
        if (mask[i] != 0.0 && ti + i*32 < D3)
            y[i] = load(Y, i*32);

    // compute dy * y and y * mask * scale
    float dyy[U];
    for (int i = 0; i < U; i++)
    {
        dyy[i] = dy[i] * y[i];
        y[i]  *= mask[i] * scale;
    }

    // reduce within thread
    for (int j = U >> 1; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
            dyy[i] = dyy[i] + dyy[i+j];
    float sum_dyy = dyy[0];

    // reduce within warp
    for (int i = 16; i > 0; i >>= 1)
        sum_dyy += shfl_xor(sum_dyy, i);

    if (blockDim.x > 32)
    {
        // first thread of each warp store to shared
        if ((tid & 31) == 0)
            Sum[tid/32] = sum_dyy;
        __syncthreads();

        if (tid < 32)
        {
            // first warp loads all prior reductions
            sum_dyy = Sum[tid];
            // reduce within this last warp
            #pragma unroll 1
            for (int i = threads_pow2/64; i > 0; i >>= 1)
                sum_dyy += shfl_xor(sum_dyy, i);
            // final reduction to shared
            Sum[tid] = sum_dyy;
        }
        __syncthreads();
        sum_dyy = Sum[0];
    }
    // dx = (dy - sum_dyy) * y * mask* scale
    DX = add_ptr_u(DX, offsetY);
    for (int i = 0; i < U; i++)
        store(DX, (dy[i] - sum_dyy) * y[i], i*32, ti + i*32 < D3);
}

// dx = (dy - sum(dy * y, axis=-1)) * y * m * scale
template <typename T>
__global__ void __launch_bounds__(32) masked_softmax_grad2(
              T*              DX,
    const     T* __restrict__ DY,
    const     T* __restrict__ Y,
    const float* __restrict__ M,
    uint D123, uint D23, uint D3, uint M1, uint M2, uint use_mask, float scale)
{
    // x: D0, D1, D2, D3
    // m:  1, D1, D2, D3
    // m:  1,  1, D2, D3
    // m:  1,  1,  1, D3
    uint tid = threadIdx.x;
    uint  d0 = blockIdx.x;
    uint  d1 = blockIdx.y;
    uint  d2 = blockIdx.z;

    uint offsetY = d0*D123 + d1*D23 + d2*D3 + tid;
    uint offsetM =           d1*M1  + d2*M2 + tid;

    // sum(dy * y, axis=-1))
    float sum_dy_y = 0.0f;
    #pragma unroll 2
    for (uint d3 = tid, offset = offsetY; d3 < D3; d3 += 32, offset += 32)
    {
        float dy = load(add_ptr_u(DY, offset));
        float  y = load(add_ptr_u(Y,  offset));
        sum_dy_y += dy * y;
    }
    for (int i = 16; i > 0; i >>= 1)
        sum_dy_y += shfl_xor(sum_dy_y, i);

    #pragma unroll 2
    for (uint d3 = tid; d3 < D3; d3 += 32, offsetY += 32, offsetM += 32)
    {
        float dy = load(add_ptr_u(DY, offsetY));
        float  y = load(add_ptr_u(Y,  offsetY));
        float  m = use_mask ? __ldg(add_ptr_u(M,  offsetM)) : 1.0f;

        float dx = (dy - sum_dy_y) * y * m * scale;

        store(add_ptr_u(DX, offsetY), dx);
    }
}

template <typename T>
bool MaskedSoftmax(CUstream stream, T* y, const T* x, const float* m, uint D0, uint D1, uint D2, uint D3, uint M1, uint M2, float scale)
{
    if (D3 > 1024*8)
        masked_softmax2<T><<<dim3(D0,D1,D2),32,0,stream>>>(y, x, m, D1*D2*D3, D2*D3, D3, M1, M2, m != NULL, scale);
    else
    {
        if (D3 > 32*4)
        {
            uint threads = CEIL_DIV(D3, 32*8) * 32;
            int thread2  = THREAD_POW2(threads);
            masked_softmax<T,8><<<dim3(D0,D1,D2),threads,0,stream>>>(y, x, m, D1*D2*D3, D2*D3, D3, M1, M2, m != NULL, scale, thread2);
        }
        else if (D3 > 32*2)
            masked_softmax<T,4><<<dim3(D0,D1,D2),32,0,stream>>>(y, x, m, D1*D2*D3, D2*D3, D3, M1, M2, m != NULL, scale,32);
        else if (D3 > 32*1)
            masked_softmax<T,2><<<dim3(D0,D1,D2),32,0,stream>>>(y, x, m, D1*D2*D3, D2*D3, D3, M1, M2, m != NULL, scale,32);
        else
            masked_softmax<T,1><<<dim3(D0,D1,D2),32,0,stream>>>(y, x, m, D1*D2*D3, D2*D3, D3, M1, M2, m != NULL, scale,32);
    }
    return true;
}

template <typename T>
bool MaskedSoftmaxGrad(CUstream stream, T* dx, const T* dy, const T* y, const float* m, uint D0, uint D1, uint D2, uint D3, uint M1, uint M2, float scale)
{
    if (D3 > 1024*4)
        masked_softmax_grad2<T><<<dim3(D0,D1,D2),32,0,stream>>>(dx, dy, y, m, D1*D2*D3, D2*D3, D3, M1, M2, m != NULL, scale);
    else
    {
        if (D3 > 32*2)
        {
            uint threads = CEIL_DIV(D3, 32*4) * 32;
            int thread2  = THREAD_POW2(threads);
            masked_softmax_grad<T,4><<<dim3(D0,D1,D2),threads,0,stream>>>(dx, dy, y, m, D1*D2*D3, D2*D3, D3, M1, M2, m != NULL, scale, thread2);
        }
        else if (D3 > 32*1)
            masked_softmax_grad<T,2><<<dim3(D0,D1,D2),32,0,stream>>>(dx, dy, y, m, D1*D2*D3, D2*D3, D3, M1, M2, m != NULL, scale,32);
        else
            masked_softmax_grad<T,1><<<dim3(D0,D1,D2),32,0,stream>>>(dx, dy, y, m, D1*D2*D3, D2*D3, D3, M1, M2, m != NULL, scale,32);
    }
    return true;
}

template bool MaskedSoftmax<float>(CUstream stream, float* y, const float* x, const float* m, uint D0, uint D1, uint D2, uint D3, uint M1, uint M2, float scale);
template bool MaskedSoftmax<ehalf>(CUstream stream, ehalf* y, const ehalf* x, const float* m, uint D0, uint D1, uint D2, uint D3, uint M1, uint M2, float scale);
template bool MaskedSoftmax<bhalf>(CUstream stream, bhalf* y, const bhalf* x, const float* m, uint D0, uint D1, uint D2, uint D3, uint M1, uint M2, float scale);

template bool MaskedSoftmaxGrad<float>(CUstream stream, float* dx, const float* dy, const float* y, const float* m, uint D0, uint D1, uint D2, uint D3, uint M1, uint M2, float scale);
template bool MaskedSoftmaxGrad<ehalf>(CUstream stream, ehalf* dx, const ehalf* dy, const ehalf* y, const float* m, uint D0, uint D1, uint D2, uint D3, uint M1, uint M2, float scale);
template bool MaskedSoftmaxGrad<bhalf>(CUstream stream, bhalf* dx, const bhalf* dy, const bhalf* y, const float* m, uint D0, uint D1, uint D2, uint D3, uint M1, uint M2, float scale);


// split_heads: (batch, pixel, head, state) -> (batch, head, pixel, state)
// merge_heads: (batch, head, pixel, state) -> (batch, pixel, head, state)
template <typename T, uint U>
__global__ void __launch_bounds__(32) transpose_0213(T* Y, const T* X, uint D123, uint D23, uint D13, uint D2, uint D3)
{
    uint  tid = threadIdx.x;
    uint  d2  = blockIdx.x;
    uint  d1  = blockIdx.y;
    uint  d0  = blockIdx.z;

    uint offset  = d0*D123 + tid;
    uint offsetX = d1*D23 + d2*D3 + offset;
    uint offsetY = d2*D13 + d1*D3 + offset;

    #pragma unroll 1
    while (d2 < D2)
    {
        #pragma unroll 1
        for (uint d3 = tid, xi = offsetX, yi = offsetY; d3 < D3; d3 += U*32, xi += U*32, yi += U*32)
        {
            const T* Xi = add_ptr_u(X, xi);
                  T* Yi = add_ptr_u(Y, yi);

            float x[U];
            for (uint i = 0; i < U; i++)
                x[i] = load(Xi, i*32, d3 + i*32 < D3);

            for (uint i = 0; i < U; i++)
                store(Yi, x[i], i*32, d3 + i*32 < D3);
        }
        offsetX += gridDim.x*D3;
        offsetY += gridDim.x*D13;
        d2      += gridDim.x;
    }
}
template <typename T>
bool Transpose_0213(CUstream stream, T* y, const T* x, uint D0, uint D1, uint D2, uint D3)
{
    // make sure each block has enough work to cover launch overhead
    uint gridX = CEIL_DIV(D2, 4);

    if (D3 <= 64)
        transpose_0213<T,2><<<dim3(gridX,D1,D0),32,0,stream>>>(y, x, D1*D2*D3, D2*D3, D1*D3, D2, D3);
    else
        transpose_0213<T,4><<<dim3(gridX,D1,D0),32,0,stream>>>(y, x, D1*D2*D3, D2*D3, D1*D3, D2, D3);
    return true;
}
template bool Transpose_0213<float>(CUstream stream, float* y, const float* x, uint D0, uint D1, uint D2, uint D3);
template bool Transpose_0213<ehalf>(CUstream stream, ehalf* y, const ehalf* x, uint D0, uint D1, uint D2, uint D3);
template bool Transpose_0213<bhalf>(CUstream stream, bhalf* y, const bhalf* x, uint D0, uint D1, uint D2, uint D3);


#endif
